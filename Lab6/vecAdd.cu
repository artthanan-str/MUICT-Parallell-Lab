
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256

__global__ void vecAdd(int *A, int *B, int *C){
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

int main(int argc, char *argv[]){
  int i;
  int size = N*sizeof(int);
  int a[N], b[N], c[N], *devA, *devB, *devC;

  for(i=0; i<N; i++){
    a[i] = 1;
    b[i] = 2;
  }

  hipMalloc( (void**) &devA, size);
  hipMalloc( (void**) &devB, size);
  hipMalloc( (void**) &devC, size);

  hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
  hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

  vecAdd<<<1, N>>>(devA, devB, devC);

  hipMemcpy( c, devC, size, hipMemcpyDeviceToHost);
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);

  for(i=0; i<N; i++){
    printf("%d", c[i]);
  }
  printf("\n");

}
