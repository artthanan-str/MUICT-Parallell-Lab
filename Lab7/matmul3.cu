
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define TILE_WIDTH 16

__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int ncols){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float PValue = 0; //PValue is used to store element of the output MatrixMulKernel
  int k = 0;

  if(row<ncols && col<ncols){
    for(k = 0; k < ncols; k++){
      float Melement = Md[row * ncols + k];
      float Nelement = Nd[k * ncols + col];
      PValue += Melement * Nelement;
    }
    Pd[row * ncols +col] = PValue;
  }
}

int main(int argc, char **argv){
  int i,j;
  int Width;

  printf("Enter Width: ");
  scanf("%d", &Width);

  int size = Width * Width * sizeof(float);
  float M[Width][Width], N[Width][Width], P[Width][Width];
  float *Md, *Nd, *Pd;
  int newValue = (Width + TILE_WIDTH -1)/TILE_WIDTH;

  for(i = 0; i < Width; i++){
    for(j = 0; j < Width; j++){
      M[i][j] = 1;
      N[i][j] = 2;
    }
  }

  hipMalloc((void**)&Md, size);
  hipMalloc((void**)&Nd, size);
  hipMalloc((void**)&Pd, size);

  hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

  //setup the execution configuration
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  dim3 dimGrid(newValue, newValue);

  //launch the device computation thread!
  MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);

  //read P from the device
  hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

  //free device matrices
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);

  for(i = 0; i < Width; i++){
    for(j = 0; j < Width; j++){
      printf("%.2f ", P[i][j]);
    }
    printf("\n");
  }
}
