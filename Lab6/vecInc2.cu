
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1000
#define M 256

__global__ void vecAdd(int *A){
  int i;
  for(i = threadIdx.x; i<N; i+=M){
    A[i] += 1;
  }
}

int main(int argc, char *argv[]){
  int i;
  int size = N*sizeof(int);
  int a[N], *devA;

  for(i=0; i<N; i++){
    a[i] = 2*i+1;
  }

  hipMalloc( (void**) &devA, size);
  hipMemcpy( devA, a, size, hipMemcpyHostToDevice);

  vecAdd<<<1, M>>>(devA);

  hipMemcpy( a, devA, size, hipMemcpyDeviceToHost);
  hipFree(devA);

  for(i=0; i<N; i++){
    if(i%26 == 0) printf("\n");
    printf("%d ", a[i]);
  }
  printf("\n");
}
