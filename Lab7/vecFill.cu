
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256
#define T 4

__global__ void vecAdd(int *A){
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < N){
    A[i] = i;
  }
}

int main(int argc, char *argv[]){
  int i;
  int blocks = N/T;
  int size = N*sizeof(int);
  int a[N], *devA;

  hipMalloc( (void**) &devA, size);

  hipMemcpy( devA, a, size, hipMemcpyHostToDevice);

  vecAdd<<<blocks, T>>>(devA);

  hipMemcpy( a, devA, size, hipMemcpyDeviceToHost);
  hipFree(devA);

  for(i=0; i<N; i++){
    if(i != 0 && i%20 == 0) printf("\n");
    printf("%d ", a[i]);
  }
  printf("\n");
}
