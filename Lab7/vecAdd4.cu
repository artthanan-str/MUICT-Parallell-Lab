
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1024
#define T 240

__global__ void vecAdd(int *A, int *B, int *C){
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < N){
    C[i] = A[i] + B[i];
  }
}

int main(int argc, char *argv[]){
  int i;
  int blocks = (N + T - 1) / T;
  int size = N*sizeof(int);
  int a[N], b[N], c[N], *devA, *devB, *devC;

  for(i=0; i<N; i++){
    a[i] = 1;
    b[i] = 2;
  }

  hipMalloc( (void**) &devA, size);
  hipMalloc( (void**) &devB, size);
  hipMalloc( (void**) &devC, size);

  hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
  hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

  vecAdd<<<blocks, T>>>(devA, devB, devC);

  hipMemcpy( c, devC, size, hipMemcpyDeviceToHost);
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);

  for(i=0; i<N; i++){
    printf("%d ", c[i]);
  }
  printf("\n");

}
