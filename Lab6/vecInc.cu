
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256

__global__ void vecAdd(int *A){
  int i = threadIdx.x;
  A[i] += 1;
}

int main(int argc, char *argv[]){
  int i;
  int size = N*sizeof(int);
  int a[N], *devA;

  for(i=0; i<N; i++){
    a[i] = 2*i+1;
  }

  hipMalloc( (void**) &devA, size);

  hipMemcpy( devA, a, size, hipMemcpyHostToDevice);

  vecAdd<<<1, N>>>(devA);

  hipMemcpy( a, devA, size, hipMemcpyDeviceToHost);
  hipFree(devA);

  for(i=0; i<N; i++){
    if(i%16 == 0) printf("\n");
    printf("%d ", a[i]);
  }
  printf("\n");

}
